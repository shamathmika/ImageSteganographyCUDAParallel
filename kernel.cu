#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <conio.h>
#include <time.h>

#define MAX_IMAGE_HEIGHT 512
#define MAX_IMAGE_WIDTH 512

__global__ void hide(int *pixels_per_process, int *text_size, char *text, unsigned char *intermediate_b, char *final_b, int *wid)
{
	int id = threadIdx.x;
	int size = blockDim.x;
	int i = id, j;
	int bits_written = 0;
	int index = id * (*text_size) / size;
	int start_index = index;
	char c = text[start_index];
	int mask = 1;
	int flag = 0;
	

	
	///*
	for (i = id * *pixels_per_process; i < (id+1) * *pixels_per_process; i++)
	{ 
	//*/
		for (j = 0; j < *wid; j++)
		{
			flag = 0;
			if (bits_written == 8)
			{
				if (index < (start_index + (*text_size) / size - 1))
				{
					c = text[++index];
					bits_written = 0;
					mask = 1;
					j = j - 1;
				}
				else
				{
					flag = 1;
					break;
				}
			}
			else
			{
				char ch = c & mask;
				int m = log((double)mask) / log((double)2);
				ch = ch >> m;
				unsigned char temp = intermediate_b[(i )* (*wid) + j] & 0xFE;
				intermediate_b[(i)* (*wid) + j] = intermediate_b[(i )* (*wid) + j] & 0xFE;
				intermediate_b[(i)* (*wid) + j] = intermediate_b[(i )* (*wid) + j] | ch;


				mask = mask << 1;
				bits_written++;
			}
		}
		///*
		if (flag == 1)
		{
			break;
		}
	}  //*/
	for (i = id*(*pixels_per_process); i < (id + 1)*(*pixels_per_process); i++)
	{
		for (j = 0; j < MAX_IMAGE_WIDTH; j++)
		{
			final_b[(i)* (*wid) + j] = intermediate_b[(i)* (*wid) + j];
		}
	}

}


/*
__global__ void  unhide(unsigned char *blue, int *wid, int *pixels_per_process, int* text_size, char *text, char *bits)
{
	int id = threadIdx.x;
	int size = blockDim.x;
	int i, j;
	int index = 8 * id * *text_size / size;
	int start_index = index;
	int ind = id * *text_size / size;
	int st = id * *pixels_per_process * *wid;
	for (i = start_index, j=0;i < start_index + (8 * *text_size / size); i++)
	{
		bits[i] = blue[st + j] & 0x01;
		j++;
	}


	int c = 0;

	for (j = 0; j < 6; j++)
	{
		c = 0;
		for (i = start_index; i < start_index + 8; i++)
		{
			c = c << 1;
			c = c | bits[j * 8 + 7 - i];
		}
		//printf("\nID:%d  c: %c", id, c);

		text[ind++] = c;
	}

}
  */



int main()
{
	float start, end;
	start = clock();

	int size = 5, text_size1, i, j, height, width, padding, pixels_per_process1;
	static unsigned char header[54], r[MAX_IMAGE_HEIGHT][MAX_IMAGE_WIDTH], g[MAX_IMAGE_HEIGHT][MAX_IMAGE_WIDTH], b[MAX_IMAGE_HEIGHT][MAX_IMAGE_WIDTH], final_r[MAX_IMAGE_HEIGHT][MAX_IMAGE_WIDTH], final_g[MAX_IMAGE_HEIGHT][MAX_IMAGE_WIDTH], final_b1[MAX_IMAGE_HEIGHT][MAX_IMAGE_WIDTH], intermediate_r[MAX_IMAGE_HEIGHT][MAX_IMAGE_WIDTH], intermediate_g[MAX_IMAGE_HEIGHT][MAX_IMAGE_WIDTH], intermediate_b1[MAX_IMAGE_HEIGHT][MAX_IMAGE_WIDTH], text1[(MAX_IMAGE_HEIGHT * MAX_IMAGE_WIDTH) / 8];

	
	

	char *image_input = "airplane.bmp";
	char *text_input = "abc.txt";
	char *image_output = "a1.bmp";
	char *text_output = "abc_out.txt";


	printf("Input image: %s\n", image_input);

	FILE *fd, *store_pixels;

	fd = fopen(image_input, "rb");

	if (fd == NULL)
	{
		printf("Error: fopen failed for %s\n", image_input);
		return 0;
	}

	store_pixels = fopen("store_input_pixels.txt", "w+");

	if (store_pixels == NULL)
	{
		printf("Error: fopen failed for store_input_pixels\n");
		return 0;
	}

	/* Read header for height, width information */

	fread(header, sizeof(unsigned char), 54, fd);

	width = *(int*)&header[18];
	height = *(int*)&header[22];
	padding = 0;

	pixels_per_process1 = height / size;


	while ((width * 3 + padding) % 4 != 0)
	{
		padding++;
	}

	printf("Dimensions of %s: %d x %d pixels\n", image_input, height, width);
	printf("Image padding: %d\n", padding);

	static unsigned char image[MAX_IMAGE_HEIGHT][MAX_IMAGE_WIDTH][3];

	for (i = 0, j = 0; i<height; j++)
	{
		fread(image[i][j], sizeof(unsigned char), 3, fd);
		fprintf(store_pixels, "image[%d][%d] = %d %d %d\n", i, j, image[i][j][0], image[i][j][1], image[i][j][2]);
		r[i][j] = image[i][j][0];
		g[i][j] = image[i][j][1];
		b[i][j] = image[i][j][2];
		if (j == (width - 1))
		{
			j = -1;
			i++;
		}
	}
	fclose(fd);

	/* Reading the text file */

	printf("Text file to be hidden: %s\n", text_input);

	FILE *f = fopen(text_input, "r");

	if (f == NULL)
	{
		printf("Error: fopen failed for %s\n", text_input);
		return 0;
	}

	fseek(f, 0, SEEK_END);
	text_size1 = ftell(f);
	fprintf(stdout, "Size of %s: %d characters\n", text_input, text_size1);
	fseek(f, 0, SEEK_SET);
	fread(text1, sizeof(unsigned char), text_size1, fd);
	text1[text_size1] = '\0';
	fprintf(stdout, "Text in %s: %s\n\n", text_input, text1);


	for (i = 0; i < MAX_IMAGE_HEIGHT; i++)
	{
		for (j = 0; j < MAX_IMAGE_WIDTH; j++)
		{
			intermediate_r[i][j] = r[i][j];
			intermediate_g[i][j] = g[i][j];
			intermediate_b1[i][j] = b[i][j];
		}
	}


	int *pixels_per_process;
	int *text_size;
	char *text;
	unsigned char *intermediate_b;
	char *final_b;
	int *wid;

	hipMalloc((void**)&pixels_per_process, sizeof(int));
	hipMalloc((void**)&text_size, sizeof(int));
	hipMalloc((void**)&text, text_size1 * sizeof(char));
	hipMalloc((void**)&intermediate_b, MAX_IMAGE_HEIGHT * MAX_IMAGE_WIDTH * sizeof(unsigned char));
	hipMalloc((void**)&final_b, MAX_IMAGE_HEIGHT * MAX_IMAGE_WIDTH * sizeof(char));
	hipMalloc((void**)&wid, sizeof(int));


	hipMemcpy(pixels_per_process, &pixels_per_process1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(text_size, &text_size1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(text, &text1, text_size1 * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(intermediate_b, intermediate_b1, MAX_IMAGE_HEIGHT * MAX_IMAGE_WIDTH * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(wid, &width, sizeof(int), hipMemcpyHostToDevice);

	hide << <1, 5 >> >(pixels_per_process, text_size, text, intermediate_b, final_b, wid);

	hipMemcpy(final_b1, final_b, MAX_IMAGE_HEIGHT * MAX_IMAGE_WIDTH * sizeof(char), hipMemcpyDeviceToHost);
	hipMemcpy(intermediate_b1, intermediate_b, MAX_IMAGE_HEIGHT * MAX_IMAGE_WIDTH * sizeof(char), hipMemcpyDeviceToHost);

	unsigned char final_image[MAX_IMAGE_HEIGHT][MAX_IMAGE_WIDTH][3];

	for (i = 0; i < height; i++)
	{
		for (j = 0; j < width; j++)
		{
			final_image[i][j][0] = intermediate_r[i][j];
			final_image[i][j][1] = intermediate_g[i][j];
			final_image[i][j][2] = intermediate_b1[i][j];
		}
	}

	printf("\nOutput image %s\n", image_output);
	printf("Dimensions of %s: %d x %d pixels\n", image_output, height, width);

	/* Write final_image into image_output */


	FILE *fd1;


	fd1 = fopen(image_output, "wb");
	if (fd1 == NULL)
	{
		printf("Error: fopen failed for %s\n", image_output);
		return 0;
	}

	fwrite(header, sizeof(unsigned char), 54, fd1);
	for (i = 0; i<height; i++)
	{
		for (j = 0; j<width; j++)
		{
			fwrite(final_image[i][j], sizeof(unsigned char), 3, fd1);
		}
	}

	fclose(fd1);

	FILE *fk;

	fk = fopen("store_output_pixels.txt", "w+");

	if (fk == NULL)
	{
		printf("Error: fopen failed for store_output_pixels.txt\n");
		return 0;
	}


	for (i = 0, j = 0; i<height; j++)
	{
		fprintf(fk, "image[%d][%d] = %d %d %d\n", i, j, final_image[i][j][0], final_image[i][j][1], final_image[i][j][2]);
		if (j == (width - 1))
		{
			j = -1;
			i++;
		}
	}
	fclose(fk);
	printf("\nDone\n");



	/* Retrieving data back */

  /*
	unsigned char blue[MAX_IMAGE_HEIGHT ][ MAX_IMAGE_WIDTH];

	for (i = 0, j = 0; i < height; j++)
	{
		blue[i][j] = final_image[i][j][2];
		if (j == (width - 1))
		{
			j = -1;
			i++;
		}
	}

	unsigned char *blue1;
	char *read_text1, read_text[MAX_IMAGE_HEIGHT * MAX_IMAGE_WIDTH];
	char *bits1;

	hipMalloc((void **)&blue1, MAX_IMAGE_HEIGHT * MAX_IMAGE_WIDTH * sizeof(unsigned char));
	hipMalloc((void **)&read_text1, text_size1 * sizeof(char));
	hipMalloc((void **)&bits1, 8 * text_size1 * sizeof(char));
	hipMemcpy(blue1, blue, MAX_IMAGE_HEIGHT * MAX_IMAGE_WIDTH * sizeof(unsigned char), hipMemcpyHostToDevice);

	unhide << <1, 5 >> > (blue1, wid, pixels_per_process, text_size, read_text, bits1);

	hipMemcpy(read_text, read_text1, text_size1 * sizeof(char), hipMemcpyDeviceToHost);

	printf("\n\nRetrieved: %s\n", read_text);
	*/

	hipFree(pixels_per_process);
	hipFree(text_size);							 
	hipFree(text);
	hipFree(intermediate_b);
	hipFree(final_b);
	hipFree(wid);
	//hipFree(blue1);

	end = clock();

	printf("\nTotal time taken: %f\n", (end - start) / CLOCKS_PER_SEC);

	return 0;
}